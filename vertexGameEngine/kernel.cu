#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "math.cuh"

vec3 projectVertex(vec3 pos, vec3 cameraPos)
{
    return { pos.x / pos.z, pos.y / pos.z, 1.0f };
}

int main()
{
    
}